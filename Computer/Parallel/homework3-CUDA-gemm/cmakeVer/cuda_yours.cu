#include "gemm.h"

void gemm_cuda_yours(float *A, float *B, float *C, int m, int k, int n,double *time_value){
    float *d_A, *d_B, *d_C;
    size_t size = m * k * sizeof(float);
    hipMalloc(&d_A, size);
    hipMemcpy(d_A, A, size,
               hipMemcpyHostToDevice);

    size = k * n * sizeof(float);
    hipMalloc(&d_B, size);
    hipMemcpy(d_B, B, size,
               hipMemcpyHostToDevice);
// Allocate C in device memory

    size = m * n * sizeof(float);
    hipMalloc(&d_C, size);


    for(int i = 0 ; i < WARMUP_TIMES ; ++i){

        ///// edit your warmup code here

        ////
    }
    timeval t1,t2;
    hipDeviceSynchronize();

    for(int i = 0 ; i < BENCH_TIMES ; ++i) {
        // hipblasSgemm('N', 'N', m, n, k, 1.0f, d_A, m, d_B, k, 0, d_C, m);

        gettimeofday(&t1, nullptr);
        ///// edit your code here



        ////

        hipDeviceSynchronize();
        gettimeofday(&t2, nullptr);
        *time_value += (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec) / 1000000.0;
    }




    hipMemcpy(C, d_C, size,
               hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}